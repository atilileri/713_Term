#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <windows.h>
#include <math.h>

#define MEM_PER_OP 32
typedef char mem_t;
const float F2_BASE = 1.02197;
const float F1_RAISE = 45.141;
const float F1_FACTOR = (3570.0 / 9999);
const unsigned int F3_FACTOR = 16;
const float F_PI = (const float)acos(-1);

//global variables for and function declerations for performance measurements
double PCFreq = 0.0;
__int64 CounterStart = 0;
void StartCounter();
double GetCounter();
void encryptDecrypt(mem_t *input, mem_t *output, mem_t *key, size_t operation_size);

int main(int argc, char *argv[]) {

	FILE *fpIn, *fpOutEnc, *fpOutDec, *fpKey;
	const size_t MEM_SIZE = sizeof(mem_t);
	mem_t key[MEM_PER_OP];
	mem_t in[MEM_PER_OP];
	mem_t outEnc[MEM_PER_OP];
	mem_t outDec[MEM_PER_OP];
	double duration = 0.0, start = 0.0, end = 0.0;

	printf("########## CPU Run ##########\n");

	fpIn = fopen("../io/input_clean_100M.bin", "rb");
	fpOutEnc = fopen("../io/outCPU_enc.bin", "wb");
	fpOutDec = fopen("../io/outCPU_dec.bin", "wb");
	fpKey = fopen("../io/key.txt", "rb");
	//start counter for performance mesaurements
	StartCounter(); 
	if (fpIn != NULL && fpKey != NULL)
	{
		printf("File Processing\n");
		size_t readKeyMemCount;
		size_t readFileMemCount;
		while(0 < (readFileMemCount = fread(in, MEM_SIZE, MEM_PER_OP, fpIn)))
		{
			//provide circular key
			readKeyMemCount = fread(key, MEM_SIZE, MEM_PER_OP, fpKey);
			if (MEM_PER_OP != readKeyMemCount)
			{
				fseek(fpKey, 0, SEEK_SET);
				readKeyMemCount += fread(&(key[readKeyMemCount]), MEM_SIZE, MEM_PER_OP - readKeyMemCount, fpKey);
			}

			//printf("Key: %.*s\n", readKeyMemCount, key);

			//printf("Read: %.*s\n", readFileMemCount, in);
			start = GetCounter();
			encryptDecrypt(in, outEnc, key, readFileMemCount);
			end = GetCounter();
			duration += (end - start);
			//printf("Encrypted: %.*s\n", readFileMemCount, outEnc);
			fwrite(outEnc, MEM_SIZE, readFileMemCount, fpOutEnc);

			start = GetCounter();
			encryptDecrypt(outEnc, outDec, key, readFileMemCount);
			end = GetCounter();
			duration += (end - start);
			//printf("Decrypted: %.*s\n", readFileMemCount, outDec);
			fwrite(outDec, MEM_SIZE, readFileMemCount, fpOutDec);
		}
	}
	else
	{
		printf("File Not Found\n");
	}
	printf("Duration of the run: %f milliseconds.\n", GetCounter());
	printf("Encrypt + Decrypt: %f milliseconds.\n", duration);
	printf("End\n");

	fclose(fpIn);
	fclose(fpOutEnc);
	fclose(fpOutDec);
	fclose(fpKey);
}

void encryptDecrypt(mem_t *input, mem_t *output, mem_t *key, size_t operation_size) {
	int i;
	for (i = 0; i < operation_size; i++) {
		mem_t cypher_key = (mem_t) abs( (int)( ( ( log(F1_FACTOR * pow(key[i]+1, F1_RAISE)) + pow(F2_BASE, key[i]+1) ) + (F3_FACTOR * sin( ((key[i] + 1) * F_PI) / 2 )) )/ 2 ) );
		output[i] = input[i] ^ cypher_key;
	}
}

void StartCounter()
{
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		printf("QueryPerformanceFrequency failed!\n");

	PCFreq = double(li.QuadPart) / 1000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}

double GetCounter()
{
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}