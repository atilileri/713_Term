#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <windows.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#define SIZE_5K (1024 * 5)				// 5KB
#define SIZE_5K_ULL4 (SIZE_5K / 32)		// 5 KB / (sizeof(unsigned long long) * 4)
#define SIZE_10M (1024 * 1024 * 10)		// 10 MB	
#define SIZE_10M_ULL4 (SIZE_10M / 32)	// 10 MB / (sizeof(unsigned long long) * 4)
#define BLOCK_SIZE 1024

typedef char mem_t;
typedef ulonglong4 memUll4_t;

const float F2_BASE = (float) 1.02197;
const float F1_RAISE = (float) 45.141;
const float F1_FACTOR = (float)(3570.0 / 9999);
const unsigned int F3_FACTOR = 16;

//global variables for and function declerations for performance measurements
double PCFreq = 0.0;
__int64 CounterStart = 0;
void StartCounter();
double GetCounter();
__global__ void encryptDecrypt(mem_t *input, mem_t *output, mem_t *key, size_t readKeySizeInBytes, size_t readFileSizeInBytes, const float F2_BASE, const float F1_RAISE, const float F1_FACTOR, const unsigned int F3_FACTOR, const float F_PI);

int main(int argc, char *argv[])
{

	//printf("%d\n", sizeof(char));						// 1
	//printf("%d\n", sizeof(unsigned int));				// 4
	//printf("%d\n", sizeof(uint4));					// 16
	//printf("%d\n", sizeof(unsigned long long));		// 8
	//printf("%d\n", sizeof(unsigned long long int));	// 8
	//printf("%d\n", sizeof(ulonglong4));				// 32
	FILE *fpIn, *fpOutEnc, *fpOutDec, *fpKey;
	const size_t MEM_ULL4_SIZE = sizeof(memUll4_t);
	memUll4_t *key, *in, *outEnc, *outDec;
	mem_t *pSrc_Dev, *pKey_Dev, *pDstEnc_Dev, *pDstDec_Dev;
	double duration = 0.0, start = 0.0, end = 0.0;

	printf("########## CUDA Run ##########\n");

	fpIn = fopen("../io/input_clean.txt", "rb");
	fpOutEnc = fopen("../io/outCUDA_enc.txt", "wb");
	fpOutDec = fopen("../io/outCUDA_dec.txt", "wb");
	fpKey = fopen("../io/key.txt", "rb");
	//start counter for performance mesaurements
	StartCounter();
	if (fpIn != NULL && fpKey != NULL && fpOutEnc != NULL && fpOutDec != NULL)
	{
		printf("Read File\n");
		size_t readKeyMemCount;
		size_t readFileMemCount;

		key = (memUll4_t*)malloc(SIZE_5K);
		in = (memUll4_t*)malloc(SIZE_10M);
		
		if ( (0 < (readFileMemCount = fread(in, MEM_ULL4_SIZE, SIZE_10M_ULL4, fpIn))) && (0 < (readKeyMemCount = fread(key, MEM_ULL4_SIZE, SIZE_5K_ULL4, fpKey))))
		{
			//printf("%d\n", readFileMemCount); //249268 which is ~(7.6M / 32)
			//printf("%d\n", readKeyMemCount); //45 which is ~(1.4K / 32)
			size_t  readFileSizeInBytes = sizeof(memUll4_t) * readFileMemCount;
			size_t  readKeySizeInBytes = sizeof(memUll4_t) * readKeyMemCount;
			outEnc = (memUll4_t*)malloc(readFileSizeInBytes);
			outDec = (memUll4_t*)malloc(readFileSizeInBytes);
			
			hipMalloc(&pSrc_Dev, readFileSizeInBytes);
			hipMalloc(&pKey_Dev, readKeySizeInBytes);

			hipMemcpy(pSrc_Dev, in, readFileSizeInBytes, hipMemcpyHostToDevice);
			hipMemcpy(pKey_Dev, key, readKeySizeInBytes, hipMemcpyHostToDevice);

			dim3 dimGrid((unsigned int)(readFileSizeInBytes / BLOCK_SIZE) + 1);
			dim3 dimBlock(BLOCK_SIZE);

			hipMalloc(&pDstEnc_Dev, readFileSizeInBytes);

			const float F_PI = (const float)acos(-1);
			start = GetCounter();
			encryptDecrypt<< <dimGrid, dimBlock >> >(pSrc_Dev, pDstEnc_Dev, pKey_Dev, readKeySizeInBytes, readFileSizeInBytes, F2_BASE ,F1_RAISE, F1_FACTOR, F3_FACTOR, F_PI);
			end = GetCounter();
			duration += (end - start);
			hipMemcpy(outEnc, pDstEnc_Dev, readFileSizeInBytes, hipMemcpyDeviceToHost);

			fwrite(outEnc, MEM_ULL4_SIZE, readFileMemCount, fpOutEnc);

			hipMalloc(&pDstDec_Dev, readFileSizeInBytes);
			start = GetCounter();
			encryptDecrypt << <dimGrid, dimBlock >> >(pDstEnc_Dev, pDstDec_Dev, pKey_Dev, readKeySizeInBytes, readFileSizeInBytes, F2_BASE, F1_RAISE, F1_FACTOR, F3_FACTOR, F_PI);
			end = GetCounter();
			duration += (end - start);
			hipMemcpy(outDec, pDstDec_Dev, readFileSizeInBytes, hipMemcpyDeviceToHost);

			fwrite(outDec, MEM_ULL4_SIZE, readFileMemCount, fpOutDec);
		}
		else
		{
			printf("Read Error\n");
		}
	}
	else
	{
		printf("File Not Found\n");
	}
	printf("Duration of the run: %f milliseconds.\n", GetCounter());
	printf("Encrypt + Decrypt: %f milliseconds.\n", duration);
	printf("End\n");

	free(key);
	free(in);
	free(outEnc);
	free(outDec);
	hipFree(pSrc_Dev);
	hipFree(pKey_Dev);
	hipFree(pDstEnc_Dev);
	hipFree(pDstDec_Dev);
	fclose(fpIn);
	fclose(fpOutEnc);
	fclose(fpOutDec);
	fclose(fpKey);

}

__global__ void encryptDecrypt(mem_t *input, mem_t *output, mem_t *key, size_t readKeySizeInBytes, size_t readFileSizeInBytes, const float F2_BASE, const float F1_RAISE, const float F1_FACTOR, const unsigned int F3_FACTOR, const float F_PI)
{
	size_t memberIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (readFileSizeInBytes > memberIndex)
	{
		int keyIndex = memberIndex % readKeySizeInBytes;
	
		mem_t cypher_key = (mem_t)abs((int)(((logf(F1_FACTOR * powf(key[keyIndex] + 1, F1_RAISE)) + powf(F2_BASE, key[keyIndex] + 1)) + (F3_FACTOR * sinf(((key[keyIndex] + 1) * F_PI) / 2))) / 2));

		output[memberIndex] = input[memberIndex] ^ cypher_key;
	}
}

void StartCounter()
{
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		printf("QueryPerformanceFrequency failed!\n");

	PCFreq = double(li.QuadPart) / 1000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}

double GetCounter()
{
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}